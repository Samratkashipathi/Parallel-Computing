#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

const short N = 100 ;

// CUDA Kernel for Convolution
__global__ void convolution ( const int *dev_a , const int *dev_b , int *dev_c)
{
      //Get the id of thread within a block
      unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x ; 
     /*for(i=0;i<m+n-1;i++){          
			y[i]=0;
            for(j=0;j<=i;j++){
			       y[i]=y[i]+(x[j]*h[i-j]);
	        }
     }*/
	  int j=0;
      if ( tid < N ) // check the boundry condition for the threads
            dev_c [tid] = 0 ;
        	for(j=0;j<=tid;j++){
        		dev_c[tid]=dev_c[tid]+dev_a[j]*dev_b[tid-j];
        	}

}

int main (void)
{

      //Host array
      int Host_a[N], Host_b[N], Host_c[N];

      //Device array
      int *dev_a , *dev_b, *dev_c ;
      time_t t;
      //Allocate the memory on the GPU
      hipMalloc((void **)&dev_a , N*sizeof(int) ) ;
      hipMalloc((void **)&dev_b , N*sizeof(int) ) ;
      hipMalloc((void **)&dev_c , N*sizeof(int) ) ;
      srand((unsigned) time(&t));

      //fill the Host array with random elements on the CPU
      for ( int i = 0; i <N ; i++ )
      {
            Host_a[i] = dev_a[i]=rand()%50;//sin(i)*sin(i); 
      }

      for ( int i = 0; i <N ; i++ )
      {
            Host_b[i] =  dev_b[i]=rand()%50;//cos(i)*cos(i); 
      }


      //Copy Host array to Device array
      hipMemcpy (dev_a , Host_a , N*sizeof(int) , hipMemcpyHostToDevice);
      hipMemcpy (dev_b , Host_b , N*sizeof(int) , hipMemcpyHostToDevice);

      //Make a call to GPU kernel
      convolution <<<(N)/512, 512>>> (dev_a , dev_b , dev_c ) ;

      //Copy back to Host array from Device array
      hipMemcpy(Host_c , dev_c , N*sizeof(int) , hipMemcpyDeviceToHost);

      //Display the result
      for ( int i = 0; i<N; i++ )
                  printf ("%d + %d = %d\n", Host_a[i] , Host_b[i] , Host_c[i] ) ;

      //Free the Device array memory
      hipFree (dev_a) ;
      hipFree (dev_b) ;
      hipFree (dev_c) ;


      return 0 ;

}
