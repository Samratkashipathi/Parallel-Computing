#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include<hip/hip_runtime.h>


__global__ void matrixMul(int A_gpu,int B_gpu,int C_gpu,int K){

    int temp = 0;
    int i=0;
    int j=0;
    
    i = blockIdx.y * blockDim.y + threadIdx.y ;   // Row i of matrix C
    j = blockIdx.x * blockDim.x + threadIdx.x  ;  // Column j of matrix C

    for(int k = 0;k<K-1;k++)
        temp+ =A_gpu[i][k] * B_gpu[k][j];

    C_gpu[i][j] = temp;
    
}



void main(){
    
    int N=16;//N=32;
    //Host array
    int A_cpu[N], B_cpu[N], C_cpu[N];

    //Device array
    int *A_gpu, *B_gpu, *C_gpu ;
    
    int K=N;

    hipMalloc((void **)&A_gpu , N*sizeof(int) ) ;
    hipMalloc((void **)&B_gpu , N*sizeof(int) ) ;
    hipMalloc((void **)&C_gpu , N*sizeof(int) ) ;


    hipMemcpy (A_gpu , A_cpu , N*sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy (B_gpu , B_cpu , N*sizeof(int) , hipMemcpyHostToDevice);

    
    dim3 dimBlock(16, 16);
    dim3 dimGrid(N/dimBlock.x, M/dimBlock.y);

    matrixMul<<<dimGrid, dimBlock>>>(A_gpu,B_gpu,C_gpu,K);
    
    hipMemcpy(C_cpu , C_gpu , N*sizeof(int) , hipMemcpyDeviceToHost);
    
    
}


