 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <math.h>

const short N = 10240 ;

// CUDA Kernel for Vector Addition
__global__ void Vector_Addition ( const int *dev_a , const int *dev_b , int *dev_c)
{
      //Get the id of thread within a block
      unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x ; 
     
      if ( tid < N ) // check the boundry condition for the threads
            dev_c [tid] = dev_a[tid] + dev_b[tid] ;

}


int main (void)
{

      //Host array
      int Host_a[N], Host_b[N], Host_c[N];

      //Device array
      int *dev_a , *dev_b, *dev_c ;

      //Allocate the memory on the GPU
      hipMalloc((void **)&dev_a , N*sizeof(int) ) ;
      hipMalloc((void **)&dev_b , N*sizeof(int) ) ;
      hipMalloc((void **)&dev_c , N*sizeof(int) ) ;

      //fill the Host array with random elements on the CPU
      for ( int i = 0; i <N ; i++ )
      {
            Host_a[i] = -i;//sin(i)*sin(i);
            Host_b[i] = i*i;//cos(i)*cos(i); 
      }

      //Copy Host array to Device array
      hipMemcpy (dev_a , Host_a , N*sizeof(int) , hipMemcpyHostToDevice);
      hipMemcpy (dev_b , Host_b , N*sizeof(int) , hipMemcpyHostToDevice);

      //Make a call to GPU kernel
      Vector_Addition <<<(N)/512, 512 >>> (dev_a , dev_b , dev_c ) ;

      //Copy back to Host array from Device array
      hipMemcpy(Host_c , dev_c , N*sizeof(int) , hipMemcpyDeviceToHost);

      //Display the result
      //for ( int i = 0; i<N; i++ )
      //            printf ("%d + %d = %d\n", Host_a[i] , Host_b[i] , Host_c[i] ) ;

      //Free the Device array memory
      hipFree (dev_a) ;
      hipFree (dev_b) ;
      hipFree (dev_c) ;


      return 0 ;

}
